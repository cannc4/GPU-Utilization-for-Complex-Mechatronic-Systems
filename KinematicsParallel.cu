#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include"Joint1.h"
#include"Joint2.h"
#include"Joint3.h"

using namespace std;

hipEvent_t start, stop;
float time1, sumTime = 0;

const float LA = 40.0;
const float LB = 68.0;
const float LA_2 = LA * LA;
const float LB_2 = LB * LB;

const float RA = 40.50;
const float RB = 30.0;
const float R = RA - RB;
const float R_2 = R * R;

const float PI = 3.141592;
const float THETA = PI * 120 / 180;

template <typename T>
class GpuMemory {
public:
    GpuMemory(int n) : size_(n*sizeof(T)) { hipMalloc(&ptr_, size_); }
    ~GpuMemory() { hipFree(ptr_); }
    operator T*() { return ptr_; }
    T* Get() { return ptr_; }
    void ReadFrom(const T* source) { hipMemcpy(ptr_, source, size_, hipMemcpyHostToDevice); }
    void WriteTo(T* dest) { hipMemcpy(dest, ptr_, size_, hipMemcpyDeviceToHost); }
    void Initialize() { hipMemset(ptr_, 0, size_); }
private:
    T* ptr_;
    int size_;
};

__global__ void ParallelPowFunction(float *nums, float *pows) {
  int INDEX = blockIdx.x * blockDim.x + threadIdx.x;  ///FOR 1D GRID 1D BLOCK
  pows[INDEX] = nums[INDEX] * nums[INDEX];
}

__global__ void InverseKinematics(float *coords /* x,y,z */,
                                  float *alphas /* alpha1,alpha2,alpha3 */,
                                  float *pows   /* x^2,y^2,z^2 */) {
    int INDEX = blockIdx.x * blockDim.x + threadIdx.x;  ///FOR 1D GRID 1D BLOCK
    /* INDEX =0   ---> x alpha1 theta1
       INDEX =1   ---> y alpha2 theta2
       INDEX =2   ---> z alpha3 theta3
       coords[0] -> x;
       coords[1] -> y;
       coords[2] -> z;
       pows[0]   -> x^2;
       pows[1]   -> y^2;
       pows[2]   -> z^2; */

    float Q = 2 * coords[0] * cos(THETA * INDEX) + 2 * coords[1] * sin(THETA * INDEX);
    float Q_2 = pow(Q,2);

    float S = (1 / LA) * (-pows[0] - pows[1] - pows[2] + LB_2 - LA_2 - R_2);///Same for All
    float S_2 = pow(S,2);///Same for All

    float T_sqrt = sqrt(4 * pows[2] + 4 * R_2 - S_2 + Q_2 * (1 - R_2 / LA_2) + Q * (-2 * R * S / LA - 4*R));
    float T = (2 * coords[2] + T_sqrt) / (-2 * R - S - Q * (R / LA - 1));

    //alphas[INDEX] = (180.0 / PI) * (-2 * atan(T)) - 30.0;
    alphas[INDEX] = -2 * atan(T); //let's calculate it in radians without offset for now
}

#define TERM_D 0
#define TERM_E 1
#define TERM_F 2
#define TERM_G 3

__global__ void ComputeDEFG(const float alphas[3], float defg[4][3]) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const float alpha = alphas[index];

    defg[0][index] = -LB_2 + LA_2 + R_2 + 2 * R * LA * cos(alpha);
    defg[1][index] = 2 * (R + LA * cos(alpha)) * cos(THETA * index);
    defg[2][index] = 2 * (R + LA * cos(alpha)) * sin(THETA * index);
    defg[3][index] = -2 * LA * sin(alpha);
}
/*
__device__ void PrintFloats(const char* name, const float *f, int count) {
    printf("%s: ", name);
    for (int i = 0; i < count; ++i)
        printf("%f, ", f[i]);
    printf("\n");
}
*/
__global__ void ComputeCoords(const float defg[4][3], float coords[3]) {
    __shared__ float products[4][3][4][3];
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    int term1 = index / 36;
    int rest = index % 36;
    int el1 = rest / 12;
    rest = rest % 12;
    int term2 = rest / 3;
    int el2 = rest % 3;
/*
#ifdef DEBUG
    if (!(term1 < 4 && el1 < 3 && term2 < 4 && el2 < 3))
        printf("OOPS. %d %d %d %d\n", term1, el1, term2, el2);
#endif
*/
    products[term1][el1][term2][el2] = defg[term1][el1] * defg[term2][el2];

    __syncthreads();

    if (index > 0)
        return;

    float h[5];
    h[0] = products[TERM_E][0][TERM_G][1] - products[TERM_E][0][TERM_G][2] -
        products[TERM_E][1][TERM_G][0] + products[TERM_E][1][TERM_G][2] +
        products[TERM_E][2][TERM_G][0] - products[TERM_E][2][TERM_G][1];
    h[1] = -(products[TERM_E][0][TERM_F][1] - products[TERM_E][0][TERM_F][2] -
        products[TERM_E][1][TERM_F][0] + products[TERM_E][1][TERM_F][2] +
        products[TERM_E][2][TERM_F][0] - products[TERM_E][2][TERM_F][1]);
    h[2] = -(products[TERM_E][0][TERM_D][1] - products[TERM_E][0][TERM_D][2] -
        products[TERM_E][1][TERM_D][0] + products[TERM_E][1][TERM_D][2] +
        products[TERM_E][2][TERM_D][0] - products[TERM_E][2][TERM_D][1]);
    h[3] = products[TERM_F][0][TERM_D][1] - products[TERM_F][0][TERM_D][2] -
        products[TERM_F][1][TERM_D][0] + products[TERM_F][1][TERM_D][2] +
        products[TERM_F][2][TERM_D][0] - products[TERM_F][2][TERM_D][1];
    h[4] = -(products[TERM_F][0][TERM_G][1] - products[TERM_F][0][TERM_G][2] -
        products[TERM_F][1][TERM_G][0] + products[TERM_F][1][TERM_G][2] +
        products[TERM_F][2][TERM_G][0] - products[TERM_F][2][TERM_G][1]);
/*
#ifdef DEBUG
    PrintFloats("D", defg[0], 3);
    PrintFloats("E", defg[1], 3);
    PrintFloats("F", defg[2], 3);
    PrintFloats("G", defg[3], 3);
    PrintFloats("H", h, 5);
#endif
*/
    float L = ((h[4]  * h[4] + h[0] * h[0]) / (h[1] * h[1])) + 1;
    float M = 2 * ((h[4] * h[3] + h[0] * h[2]) / (h[1] * h[1])) -
        ((h[4] * defg[TERM_E][0] + h[0] * defg[TERM_F][0]) / h[1]) - defg[TERM_G][0];
    float Q = ((h[3] * h[3] + h[2] * h[2]) / (h[1] * h[1])) -
        ((h[3] * defg[TERM_E][0] + h[2] * defg[TERM_F][0]) / h[1]) + defg[TERM_D][0];
/*
#ifdef DEBUG
    float LMQ[3] = {L,M,Q};
    PrintFloats("LMQ", LMQ, 3);
#endif
*/
    coords[2] = (-M - sqrt(M * M - 4 * L * Q)) / (2 * L);

    coords[0] = (coords[2] * h[4]) / h[1] + h[3] / h[1];
    coords[1] = (coords[2] * h[0]) / h[1] + h[2] / h[1];
    coords[2] = coords[2]; // +nozzle will be added
}

void ForwardKinematics(const float alphas[3], float coords[3]) {
    GpuMemory<float> d_alphas(3);
    d_alphas.ReadFrom(alphas);
    GpuMemory<float> d_defg(4 * 3);
    ComputeDEFG <<< 1, 3 >>>(d_alphas, reinterpret_cast<float (*)[3]>(d_defg.Get()));

    GpuMemory<float> d_coords(3);
    ComputeCoords <<< 1, 4 * 3 * 4 * 3 >>>(reinterpret_cast<float (*)[3]>(d_defg.Get()), d_coords);
    d_coords.WriteTo(coords);
}

__global__ void MaptoEncoder(float *alpha) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    float pulse = 360.0f / 4000;
    int temp = (alpha[index]/pulse);
    alpha[index] = temp * pulse;
}

int main() {

    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Change these values to compute with different inputs.
    float coords[3] = { 4.4239, 2.60742, -54.1189 }; // x, y, z
    float alphas[3] = { 0.1, 0.2, 0.3 }; // alpha1 , alpha2, alpha3

    GpuMemory<float> coords_D(3);
    GpuMemory<float> alphas_D(3);
    GpuMemory<float> pows_D(3);
    
    ofstream XYZoutput("ParallelXYZoutput.txt");
    XYZoutput << "X\tY\tZ" << endl;
    ofstream AlphaOutput("ParallelAlphaOutput.txt");
    AlphaOutput << "Alpha1\tAlpha2\tAlpha3" << endl;
    
    for(int i=0; i<sizeOfArray; ++i)
    {
    	    alphas[0] = Joint1Angle[i];
    	    alphas[1] = Joint2Angle[i];
    	    alphas[2] = Joint3Angle[i];
    	    	
	    hipEventRecord(start, 0);
	    
	    alphas_D.ReadFrom(alphas);
	    
	    float coords_out[3];
	    ForwardKinematics(alphas_D, coords_out);
	    
	    coords_D.ReadFrom(coords_out);

	    ParallelPowFunction <<< dim3(1,1,1), dim3(3,1,1) >>> (coords_D, pows_D);
	    
	    float pows[3];   // x^2, y^2, z^2
	    pows_D.WriteTo(pows);

	    InverseKinematics <<< dim3(1,1,1), dim3(3,1,1) >>> (coords_D, alphas_D, pows_D);
	    float alpha_out[3];
	    alphas_D.WriteTo(alpha_out);
	    
	    hipEventRecord(stop, 0);
	    hipEventSynchronize(stop);
	    hipEventElapsedTime(&time1, start, stop);
		
	    sumTime += time1;

	    XYZoutput << coords_out[0] << " " << coords_out[1] << " " << coords_out[2] << endl; 
	    AlphaOutput << alpha_out[0] << " " << alpha_out[1] << " " << alpha_out[2] << endl;

	/*
	    std::cout << "Input coords: " << coords[0] << ", " << coords[1] << ", " << coords[2] << "\n";
	    std::cout << "  inverse kinematics result angles: " << alpha_out[0] << ", " << alpha_out[1] << ", "
		<< alpha_out[2] << "\n";
	    std::cout << "Input alphas: " << alphas[0] << ", " << alphas[1] << ", " << alphas[2] << "\n";
	    std::cout << "  forward kinematics result coords: " << coords_out[0] << ", " << coords_out[1] << ", "
		<< coords_out[2] << "\n";
    	*/
    }
    
    cout << sumTime << endl;

    XYZoutput.close();
    AlphaOutput.close();    
    return 0;
}

/* OUTPUT:
 * Input coords: 4.4239, 2.60742, -54.1189
 *   inverse kinematics result angles: 0.100001, 0.200001, 0.3
 * Input alphas: 0.1, 0.2, 0.3
 *   forward kinematics result coords: 4.4239, 2.60742, -54.1189
 */


